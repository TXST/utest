#include <hip/hip_runtime.h>

#include <stdio.h>

#define N  1024

__global__ void vector_dot_product(float *a, float *b, float *c) {

  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < N) {
    c[i] = a[i] * b[i];
  }

}

int main() {

  float a[N], b[N], c[N];
  
    printf("Hello ztb!\n");

  // 初始化 a 和 b
  for (int i = 0; i < N; i++) {
    a[i] = i;
    b[i] = i + 1;
  }


float *dev_a, *dev_b, *dev_c;

  hipMalloc(&dev_a, N * sizeof(float));
  hipMalloc(&dev_b, N * sizeof(float));
  hipMalloc(&dev_c, N * sizeof(float));

  hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, N * sizeof(float), hipMemcpyHostToDevice);


  int threadsPerBlock = 256;
  int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
  vector_dot_product<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_c);

  hipDeviceSynchronize();

  hipMemcpy(c, dev_c, N * sizeof(float), hipMemcpyDeviceToHost);

  // 打印结果
  for (int i = 0; i < N; i++) {
    printf("c[%d] = %f\n", i, c[i]);
  }

  hipFree(a);
  hipFree(b);
  hipFree(c);

        hipDeviceReset();//重置CUDA设备释放程序占用的资源
    system("pause");

  return 0;
}